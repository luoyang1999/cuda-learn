
#include <hip/hip_runtime.h>
#include<stdio.h>

// SAXPY stands for “Single-precision A*X Plus Y”
__global__ void saxpy(int n, float a, float *x, float *y) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < n)
        y[id] = a * x[id] + y[id];
}


int main() {
    int N = 1 << 20;
    float *x, *y, *d_x, *d_y;
    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for(int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block_size = 256;
    dim3 grid_size = (N + 255) / 256;
    
    hipEventRecord(start);
    saxpy<<<grid_size, block_size>>>(N, 2.0f, d_x, d_y);
    hipEventRecord(stop);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Ms: %f\n", milliseconds);
    printf("effective Bandwidth(gb/s): %f\n", N*4*3/milliseconds/1e6);

    float maxError = 0.0f;
    for (int i=0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 4.0f));
        if (maxError > 0.1f) {
            printf("%f %f\n", x[i], y[i]);
            break;
        }
    }

    printf("max error: %.2f\n", maxError);

    free(x);
    free(y);
    hipFree(x);
    hipFree(y);

}