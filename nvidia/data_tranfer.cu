
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>

inline hipError_t checkCuda(hipError_t result) {
# if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
# endif
    return result;
}

void profileCopies(float *h_a, float *h_b, float *d, unsigned int n, const char *desc) {
    printf("\n%s transfers\n", desc);

    unsigned int bytes = n * sizeof(float);

    hipEvent_t start, stop;
    checkCuda(hipEventCreate(&start));
    checkCuda(hipEventCreate(&stop));

    printf("Host to Device\n");
    checkCuda(hipEventRecord(start, 0));
    checkCuda(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
    checkCuda(hipEventRecord(stop, 0));
    checkCuda(hipEventSynchronize(stop));
    float time;
    checkCuda(hipEventElapsedTime(&time, start, stop));
    printf("Time: %f ms\n", time);

    printf("Device to Host\n");
    checkCuda(hipEventRecord(start, 0));
    checkCuda(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
    checkCuda(hipEventRecord(stop, 0));
    checkCuda(hipEventSynchronize(stop));
    checkCuda(hipEventElapsedTime(&time, start, stop));
    printf("Time: %f ms\n", time);

    for (int i = 0; i < n; i++) {
        if (h_a[i] != h_b[i]) {
            printf("mismatch at %d: %f != %f\n", i, h_a[i], h_b[i]);
            break;
        }
    }

    checkCuda(hipEventDestroy(start));
    checkCuda(hipEventDestroy(stop));
}

int main() {
    // 0 测量传输时间
    // const unsigned int N = 1048576; // 2^20 1mB
    // const unsigned int bytes = N * sizeof(int);

    // int *h_a = (int *)malloc(bytes);
    // memset(h_a, 0, bytes);

    // int *d_a;
    // cudaMalloc(&d_a, bytes);

    // cudaMemcpy(d_a, h_a, bytes, cudaMemcpyHostToDevice);
    // cudaMemcpy(h_a, d_a, bytes, cudaMemcpyDeviceToHost);

    // 1 使用固定内存
    unsigned int nElements = 4 * 1024 * 1024;
    const unsigned int bytes = nElements * sizeof(float);

    // host arrays
    float *h_aPageable, *h_bPageable;   // 普通的内存
    float *h_aPinned, *h_bPinned;       // 固定内存

    // device array
    float *d_a;

    // allocate and initialize
    h_aPageable = (float *)malloc(bytes);
    h_bPageable = (float *)malloc(bytes);
    checkCuda(hipHostMalloc((void **)&h_aPinned, bytes, hipHostMallocDefault));
    checkCuda(hipHostMalloc((void **)&h_bPinned, bytes, hipHostMallocDefault));
    checkCuda(hipMalloc((void **)&d_a, bytes));
    // output device info and transfer size
    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, 0) );

    printf("\nDevice: %s\n", prop.name);
    printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));

    // perform copies and report bandwidth
    profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
    profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");

    hipFree(d_a);
    hipHostFree(h_aPinned);
    hipHostFree(h_bPinned);
    free(h_aPageable);
    free(h_bPageable);
}