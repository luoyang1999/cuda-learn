#include<stdio.h>
#include"../tool/common.cuh"

int main() {
    int *ipHost_A;
    ipHost_A = (int*)malloc(sizeof(int));

    memset(ipHost_A, 0, sizeof(int));

    int *ipDev_A;
    ErrorCheck(hipMalloc((int**)&ipDev_A, sizeof(int)), __FILE__, __LINE__);

    ErrorCheck(hipMemcpy(ipHost_A, ipDev_A, sizeof(int), hipMemcpyHostToDevice), __FILE__, __LINE__);

    free(ipHost_A);
    ErrorCheck(hipFree(ipDev_A), __FILE__, __LINE__);
}