
#include <hip/hip_runtime.h>
#include<stdio.h>

int main() {
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);

    if (error != hipSuccess || iDeviceCount == 0) {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        return 1;
    }
    printf("Device count: %d\n", iDeviceCount);

    // 设置执行
    int iDev = 0;
    error = hipSetDevice(iDev);
    if (error != hipSuccess) {
        printf("hipSetDevice returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        return 1;
    }
    printf("Set device: %d\n", iDev);
}