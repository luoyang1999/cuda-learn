
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void add(int* a, int* b, int* c, int num)
{
    // 获取当前线程的id
    int i = threadIdx.x;
    if(i < num)
        c[i] = a[i] + b[i];
}

int main(void)
{
    // init data
    int num = 10;
    int a[num], b[num], c[num];
    int *a_gpu, *b_gpu, *c_gpu;

    for(int i=0; i<num;i++)
    {
        a[i] = i;
        b[i] = i * i;
    }

    hipMalloc((void **)&a_gpu, num * sizeof(int));
    hipMalloc((void **)&b_gpu, num * sizeof(int));
    hipMalloc((void **)&c_gpu, num * sizeof(int));

    // copy data
    hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);

    // do
    // 几块几个线程，最多也就512个线程
    // block 可能可以上千
    add<<<1, num>>>(a_gpu, b_gpu, c_gpu, num);

    // get
    hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

    // visualization
    for(int i=0;i<num;i++){
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}